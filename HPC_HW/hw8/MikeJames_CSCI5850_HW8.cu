// Homework 8: CUDA implementation
// Mike James
// 5/3/2018


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#define max 1024
#define elements 2

__global__ void dotprod(float *x, float *y, float *k, int *i) {
	float sum = 0.0;
	for (int m = 0; m < *i; m++) {
		sum = x[m] * y[m];
		__syncthreads();
		*k = *k + sum;
	}
}

int main(int argc, char *argv[]) 
{
	// Host variables
	unsigned int N[elements] = {100, 1024};
	float x[max], y[max];
	int i = 0, j = 0, intSize = 0;
	float k = 0.0, size = 0.0; 
	
	// Calculate the total size of "size"
	size = sizeof(float);
	intSize = sizeof(int);
	
	// GPU variables
	float *d_x, *d_y, *d_k;
	int *d_i;

	// Adjusting the size
	i = N[atoi(argv[1])];	// Setting "i" equal to N[x] based on the CLA (0 or 1)
	size = i * size;		// Recalculating "size" by taking the size of a float times the number of items in N[x]
	intSize = i * intSize;	// Recalculating "size" "  "      "   "    "  " int   "     "   "      "  "     "  "
	
	// initialize data (x and y)
	for (j = 0; j < max; j++) {
		x[j] = y[j] = 1.0;
		//x[j] = 1.0;
		//if (j%2 == 0) {
		//	y[j] = 1.0;
		//}
		//else {
		//	y[j] = 2.0;
		//}
	}
		
	// GPU variable allocation
	hipMalloc((void **)&d_x, size); 
	hipMalloc((void **)&d_y, size); 
	hipMalloc((void **)&d_k, size);
	hipMalloc((void **)&d_i, intSize);
	
	// send partial data to GPU
	hipMemcpy(d_x, &x, size, hipMemcpyHostToDevice); 
	hipMemcpy(d_y, &y, size, hipMemcpyHostToDevice);
	hipMemcpy(d_i, &i, intSize, hipMemcpyHostToDevice);

	// Launching kernel on GPU
	dotprod<<<1,i>>>(d_x, d_y, d_k, d_i); 
	
	// Collect everything back to Host
	hipMemcpy(&k, d_k, size, hipMemcpyDeviceToHost); 
	
	// Answer
	printf ("k = %f\n\n", k);
	
	// Clean up after CUDA
	hipFree(d_x); 
	hipFree(d_y); 
	hipFree(d_k); 
	hipFree(d_i);

	return 0;
}
